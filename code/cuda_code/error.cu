#include <hip/hip_runtime.h>
#include <stdio.h>

void check(hipError_t err, const char* const func, const char* const file,
           const int line)
{
    if (err != hipSuccess)
    {
        printf("CUDA Runtime Error at: %s:%d\n", file, line);
        // 
        printf("%s %s\n", hipGetErrorString(err), func);
    }
}

void CHECK_CUDA_ERROR(hipError_t val)
{
    check(val, "error", __FILE__, __LINE__);
}